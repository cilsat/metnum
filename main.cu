#include "matrix.cu"
#include <string.h>

#define GRID_SIZE 100
#define BLOCK_SIZE 512
#define MATRIX_SIZE 10

int main(int argc, char **argv) {
    // Initialize CUDA device
    printf("[MetNum CUDA Routines]\n");
    hipDeviceProp_t deviceProp;
    hipSetDevice(0);
    hipGetDeviceProperties(&deviceProp, 0);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", 0, deviceProp.name, deviceProp.major, deviceProp.minor);

    // Initialize blocks and dimensions
    long n = MATRIX_SIZE;
    long msize = n*n*sizeof(double);
    long vsize = n*sizeof(double);

    double *hA = (double *)malloc(msize);
    double *hb = (double *)malloc(vsize);
    double *hc = (double *)malloc(vsize);
    double *dA, *db, *dc;

    hipMalloc((void **) &dA, msize);
    hipMalloc(&db, vsize);
    hipMalloc(&dc, vsize);

    hipMemcpy(dA, hA, msize, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, vsize, hipMemcpyHostToDevice);
    hipMemcpy(dc, hc, vsize, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*
    h = m_init(n, n);
    hipMallocPitch(&d, &pitch, n*sizeof(double), n);
    hipMalloc(&dest, n*n*sizeof(double));

    cu_hilbert<<< GRID_SIZE, BLOCK_SIZE >>>(d, dest, pitch, n, n);
    hipMemcpy(h->data, dest, n*n*sizeof(double), hipMemcpyDeviceToHost);
    m_print(h);

    m_del(h);
    hipFree(d);
    hipFree(dest);
    */

    hipFree(dA);
    hipFree(db);
    hipFree(dc);
    free(hA);
    free(hb);
    free(hc);
    return 0;
}
